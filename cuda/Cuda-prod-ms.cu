#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>   //for CUDA runtime API
#include <hip/hip_runtime_api.h>    //for checkCudaError macro
#include <helper_timer.h>   //for CUDA SDK timers

#define BD 1024   //x-dimension of thread blocks



//simple CPU implementation of matrix-matrix product
void cpuMatrixProduct(int m, int k, int n, const float *A, const float *B, float *C) {
    //auxiliary variables
    int index_m;
    int index_k;
    int index_n;

    for(index_m=0; index_m<m; index_m++) {  //TODO: check which is the best order of the three nested loops.
        for(index_k=0; index_k<k; index_k++) {
            for(index_n=0; index_n<n; index_n++) {
                C[index_m*n + index_n] += A[index_m*k + index_k] * B[index_k*n + index_n];

            }

        }

    }

}



//GPU implementation of matrix-matrix product
//In this version, we use a block of threads foreach matrix C component and we divide A-rows and B-columns between multiple threads belonging to the same block.
__global__ void gpuMatrixProduct(int m, int k, int n, const float *A, const float *B, float *C) {
    //auxiiary variables
    int index_k;    //index_k depends on tid (threadIdx.x); we loop on index_k in order to iterate on a single A-row and on a single B-column.
    int tid = threadIdx.x;
    int c_row = blockIdx.y;
    int c_col = blockIdx.x;
    float t = 0.0;  //partial result of matrix matrix product
    if(c_row >= m || c_col >= n)  return; //case in which thread indexes exceed matrix C dimensions

    //use of shared memory
    extern __shared__ float aux[BD];
    //matrix matrix product
    for(index_k=tid; index_k<k; index_k += blockDim.x) { 
        t += A[index_k+c_row*k] * B[c_col+index_k*n];
    }
    aux[tid] = t;
    __syncthreads();

    //reduction + write result to global memory
    //TODO: SI PUO' VELOCIZZARE
    for(unsigned int s=1; s < blockDim.x; s*=2) {
        int index = 2*s*tid;
        if(index < blockDim.x)
            aux[index] += aux[index+s];
        __syncthreads();

    }

    //write result to global memory
    if(tid == 0) C[c_col+c_row*n] += aux[0];

}



int main(int argc, char **argv) {
    //auxiliary variables
    int row;    //row & col are used as indexes loop
    int col;
    int idx;    //matrix index (= row*ncols + col)

    if(argc < 5) {
        fprintf(stderr, "Usage: %s m k n exec_on_cpu\n", argv[0]);
        return -1;
    }

    int m = atoi(argv[1]);
    int k = atoi(argv[2]);
    int n = atoi(argv[3]);
    char *exec_cpu = argv[4];

    //HOST MEMORY INITIALIZATION
    float *h_A = new float[m*k];    //matrix A
    float *h_B = new float[k*n];    //matrix B
    float *h_C = new float[m*n];    //matrix C
    float *h_C_d = new float[m*n];  //output (matrix C) copied from device memory

    srand(123456);  //seed
    for(row=0; row<m; row++) {  //matrix A initialization
        for(col=0; col<k; col++) {
            idx = row*k + col;
            //h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
            h_A[idx] = 1.0*(idx%10);

        }

    }
    for(row=0; row<k; row++) {  //matrix B initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            //h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
            h_B[idx] = 2.0*(idx%10);

        }

    }
    for(row=0; row<m; row++) {  //matrix C initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            //h_C[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
            h_C[idx] = 1.0;

        }

    }

    std::cout << "Test case: m=" << m << ", k=" << k << ", n=" << n << std::endl;

    //DEVICE MEMORY INITIALIZATION
    float *d_A; //matrix A
    float *d_B; //matrix B
    float *d_C; //matrix C

    checkCudaErrors(hipMalloc((void **) &d_A, m*k*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_B, k*n*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_C, m*n*sizeof(float)));

    //copy matrices from the host (CPU) to the device (GPU)
    checkCudaErrors(hipMemcpy(d_A, h_A, m*k*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, k*n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, m*n*sizeof(float), hipMemcpyHostToDevice));

    //CALCULATIONS ON THE CPU - it is useful to check if the calculations on the GPU are correctly made through a comparison of the results.
    float flopCnt = 2.e-6*m*k*n;    //in this case, FLOPS = 2*m*k*n/TIME
    float cpuFlops;
    float gpuFlops;

    //Create the CUDA SDK timer
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    if(exec_cpu[0] == 'y') {
        timer->start();
        cpuMatrixProduct(m, k, n, h_A, h_B, h_C);
        timer->stop();

        cpuFlops = flopCnt / timer->getTime();
        std::cout << "CPU time: " << timer->getTime() << " ms.  GFLOPS: " << cpuFlops << std::endl;
        timer->reset();
    }

    //CALCULATIONS ON THE GPU
    const dim3 BLOCK_DIM(BD);
    const dim3 GRID_DIM(n, m); //this way we have one thread-block foreach matrix C component.

    timer->start();
    gpuMatrixProduct<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_C);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    gpuFlops = flopCnt / timer->getTime();
    std::cout << "GPU time: " << timer->getTime() << " ms.  GFLOPS: " << gpuFlops << std::endl;

    if(exec_cpu[0] == 'y') {
        //download the resulting matrix d_C from the device and store it in h_C_d.
        checkCudaErrors(hipMemcpy(h_C_d, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost));

        //now let's check if the results are the same.
        float relativeDiff = 0.0f;
        float diff = 0.0f;
        float maxAbs;
        int errCount = 0;

        for(row=0; row<m; row++) {  //comparison between every single entry of h_C with every single entry of h_C_d.
            for(col=0; col<n; col++) {
                idx = row*n + col;

                maxAbs = std::max(std::abs(h_C[idx]), std::abs(h_C_d[idx]));
                if(maxAbs == 0.0)
                    maxAbs = 1.0;
                relativeDiff = std::max(relativeDiff, std:: abs(h_C[idx] - h_C_d[idx])/maxAbs);
                diff = std::max(diff, std::abs(h_C[idx] - h_C_d[idx]));

                if(relativeDiff > 0.001)
                    errCount++;

            }

        }
        //relativeDiff should be as close as possible to unit roundoff.
        //float corresponds to IEEE single precision, so unit roundoff is 1.19e-07.
        std::cout << "Max diff = " << diff << ";    Max relative diff = " << relativeDiff << std::endl;
        std::cout << "Err count = " << errCount << std::endl;

        /*std::cout << "" << std::endl;
        for(idx=0; idx<m*k; idx++) {
            std::cout << "h_A[" << idx/k << "][" << idx%k << "] = " << h_A[idx] << std::endl;
        }

        std::cout << "" << std::endl;
        for(idx=0; idx<n*k; idx++) {
            std::cout << "h_B[" << idx/n << "][" << idx%n << "] = " << h_B[idx] << std::endl;
        }

        for(idx=0; idx<m*n; idx++) {
            if(h_C[idx] - h_C_d[idx] > 0.1 || h_C[idx] - h_C_d[idx] < -0.1) {
                printf("\nh_C[%d][%d] = %f\n", idx/n, idx%n, h_C[idx]);
                printf("h_C_d[%d][%d] = %f\n", idx/n, idx%n, h_C_d[idx]);
            }
        }*/
    }
    

    //CLEANING UP
    delete timer;
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_d;

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    return 0;

}
