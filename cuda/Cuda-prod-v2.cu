#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>   //for CUDA runtime API
#include <hip/hip_runtime_api.h>    //for checkCudaError macro
#include <helper_timer.h>   //for CUDA SDK timers

//#define XBD 512 //x-dimension of thread blocks
//#define YBD 2   //y-dimension of thread blocks

//simple CPU implementation of matrix-matrix product
void cpuMatrixProduct(int m, int k, int n, const float *A, const float *B, float *C) {
    //auxiliary variables
    int index_m;
    int index_k;
    int index_n;
    for(index_m=0; index_m<m; index_m++) {
        for(index_k=0; index_k<k; index_k++) {
            for(index_n=0; index_n<n; index_n++) {
                C[index_m*n + index_n] += A[index_m*k + index_k] * B[index_k*n + index_n];
            }
        }
    }
}



//GPU implementation of matrix-matrix product
//In this version, we use a block of threads for each block of rows of matrix C (and of matrix A) --> all threads work on entire matrix B.
__global__ void gpuMatrixProduct(int m, int k, int n, const float *A, const float *B, float *C) {
    //auxiiary variables
    int index_k;    //index_k is a pure loop index.
    int idx_2;
    int shift;
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int row = tid_y + blockIdx.x * blockDim.y;
    if(row >= m || tid_x >= n)  return; //case in which thread indexes exceed matrix C dimensions

    //matrix matrix product
    for(; tid_x<n; tid_x += blockDim.x) {
        int num_col=tid_x/32;
        int rest_col=tid_x%32;
        idx_2=num_col*k*32+rest_col;
        if(tid_x/32<(n/32))
            shift=32;
        else shift=n%32;
        for(index_k=0; index_k<k; index_k++) {
            //C[tid_x+row*n] += A[index_k+row*k] * B[tid_x+index_k*n];
            C[tid_x+row*n] += A[index_k+row*k] * B[idx_2];
            idx_2+=shift;
        }
    }
}



int main(int argc, char **argv) {
    //auxiliary variables
    int row;    //row & col are used as indexes loop
    int col;
    int idx;    //matrix index (= row*ncols + col)

    if(argc < 7) {
        fprintf(stderr, "Usage: %s m k n exec_on_cpu XBD YBD\n", argv[0]);
        return -1;
    }

    int m = atoi(argv[1]);
    int k = atoi(argv[2]);
    int n = atoi(argv[3]);
    char *exec_cpu = argv[4];

    int XBD = atoi(argv[5]);
    int YBD = atoi(argv[6]);

    //HOST MEMORY INITIALIZATION
    float *h_A = new float[m*k];    //matrix A
    float *h_B = new float[k*n];    //matrix B on CPU
    float *h_BG = new float[k*n];   //matrix B on GPU
    float *h_C = new float[m*n];    //matrix C
    float *h_C_d = new float[m*n];  //output (matrix C) copied from device memory

    srand(123456);  //seed
    for(row=0; row<m; row++) {  //matrix A initialization
        for(col=0; col<k; col++) {
            idx = row*k + col;
            h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;

        }

    }
    int pos_colonna;
    int rest_colona;
    int idx_2;
    for(row=0; row<k; row++) {  //matrix B initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
            
            pos_colonna=col/32;
            rest_colona=col%32;
            if(pos_colonna < (n/32))
                idx_2 = pos_colonna*32*k + row*32 + rest_colona;
            else
                idx_2 = pos_colonna*32*k + row * (n % 32) + rest_colona;
            h_BG[idx_2] =h_B[idx];
        }

    }
    for(row=0; row<m; row++) {  //matrix C initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            h_C[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;

        }

    }

    //std::cout << "Test case: m=" << m << ", k=" << k << ", n=" << n << std::endl;

    //DEVICE MEMORY INITIALIZATION
    float *d_A; //matrix A
    float *d_B; //matrix B
    float *d_C; //matrix C

    checkCudaErrors(hipMalloc((void **) &d_A, m*k*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_B, k*n*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_C, m*n*sizeof(float)));

    //copy matrices from the host (CPU) to the device (GPU)
    checkCudaErrors(hipMemcpy(d_A, h_A, m*k*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_BG, k*n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, m*n*sizeof(float), hipMemcpyHostToDevice));
    //CALCULATIONS ON THE CPU - it is useful to check if the calculations on the GPU are correctly made through a comparison of the results.
    float flopCnt = 2.e-6*m*k*n;    //in this case, FLOPS = 2*m*k*n/TIME
    float cpuFlops;
    float gpuFlops;

    //Create the CUDA SDK timer
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    if(exec_cpu[0] == 'y') {
        timer->start();
        cpuMatrixProduct(m, k, n, h_A, h_B, h_C);
        timer->stop();

        cpuFlops = flopCnt / timer->getTime();
        std::cout << "CPU time: " << timer->getTime() << " ms.  GFLOPS: " << cpuFlops << std::endl;
        timer->reset();
    }

    //CALCULATIONS ON THE GPU
    const dim3 BLOCK_DIM(XBD, YBD);
    const dim3 GRID_DIM((m-1+YBD)/YBD); //this way we have the right number of block rows even if m is not multiple of YBD.

    timer->start();
    gpuMatrixProduct<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_C);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    gpuFlops = flopCnt / timer->getTime();
    std::cout << "\"GPU_time\": "<< timer->getTime() << ",\"GFLOPS\":" << gpuFlops <<std::endl;

    if(exec_cpu[0] == 'y') {
        //download the resulting matrix d_C from the device and store it in h_C_d.
        checkCudaErrors(hipMemcpy(h_C_d, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost));

        //now let's check if the results are the same.
        float relativeDiff = 0.0f;
        float diff = 0.0f;
        float maxAbs;

        for(row=0; row<m; row++) {  //comparison between every single entry of h_C with every single entry of h_C_d.
            for(col=0; col<n; col++) {
                idx = row*n + col;
                maxAbs = std::max(std::abs(h_C[idx]), std::abs(h_C_d[idx]));
                if(maxAbs == 0.0)
                    maxAbs = 1.0;
                relativeDiff = std::max(relativeDiff, std:: abs(h_C[idx] - h_C_d[idx])/maxAbs);
                diff = std::max(diff, std::abs(h_C[idx] - h_C_d[idx]));

            }

        }
        //relativeDiff should be as close as possible to unit roundoff.
        //float corresponds to IEEE single precision, so unit roundoff is 1.19e-07.
        std::cout << "Max diff = " << diff << ";    Max relative diff = " << relativeDiff << std::endl;
    }

    //CLEANING UP
    delete timer;
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_d;

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    return 0;

}
