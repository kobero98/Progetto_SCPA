#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>   //for CUDA runtime API
#include <hip/hip_runtime_api.h>    //for checkCudaError macro
#include <helper_timer.h>   //for CUDA SDK timers
#include <mma.h>

using namespace nvcuda;

#define BD 1024   //x-dimension of thread blocks

const int TILE_WIDTH = 32;


//simple CPU implementation of matrix-matrix product
void cpuMatrixProduct(int m, int k, int n, const float *A, const float *B, float *C) {
    //auxiliary variables
    int index_m;
    int index_k;
    int index_n;

    for(index_m=0; index_m<m; index_m++) {
        for(index_k=0; index_k<k; index_k++) {
            for(index_n=0; index_n<n; index_n++) {
                C[index_m*n + index_n] += A[index_m*k + index_k] * B[index_k*n + index_n];
            }
        }
    }
}
__global__ void matrixMulti(float* A_d, float* B_d, float* C_d, int m, int k, int n)
{
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int col = blockIdx.x*blockDim.x + threadIdx.x; //la colonna di mio interesse
    int row = blockIdx.y*blockDim.y + threadIdx.y; //la riga di mio interesse

    int tx = threadIdx.x; //dove devo lavorare io sulla matrice ausiliaria aka i
    int ty = threadIdx.y; //dove devo lavorare io sulla matrice ausiliaria aka j
    float sum = 0.0;

    for(int t=0; t<(k-1)/TILE_WIDTH+1; t++)
    {
        if(row<m && t*TILE_WIDTH+tx<k)
            ds_A[ty][tx] = A_d[row*k + t*TILE_WIDTH+tx]; //e se cambiassi la disposizione della matrice A?
            //e se un thread mettesse più dati in memoria condivisa? 
            //in modo tale da non pagare il costo dello swap out dei thread per una singola istruzione
        else
            ds_A[ty][tx] = 0.0;
        if(t*TILE_WIDTH+ty<k && col<n)
            ds_B[ty][tx] = B_d[(t*TILE_WIDTH+ty)*n + col]; //e se cambiassi la disposizione della matrice B?
        else
            ds_B[ty][tx] = 0.0;
        __syncthreads();
        for(int i=0; i<TILE_WIDTH; i++)
            sum += ds_A[ty][i] * ds_B[i][tx];
        __syncthreads();
    }
    if(row<m && col<n)
        C_d[col+row*n] += sum;
}



int main(int argc, char **argv) {
    //auxiliary variables
    int row;    //row & col are used as indexes loop
    int col;
    int idx;    //matrix index (= row*ncols + col)

    if(argc < 5) {
        fprintf(stderr, "Usage: %s m k n exec_on_cpu\n", argv[0]);
        return -1;
    }
    int m = atoi(argv[1]);
    int k = atoi(argv[2]);
    int n = atoi(argv[3]);
    char *exec_cpu = argv[4];
    
    //HOST MEMORY INITIALIZATION
    float *h_A = new float[m*k];    //matrix A
    float *h_B = new float[k*n];    //matrix B
    float *h_C = new float[m*n];    //matrix C
    float *h_C_d = new float[m*n];  //output (matrix C) copied from device memory

    srand(123456);  //seed
    for(row=0; row<m; row++) {  //matrix A initialization
        for(col=0; col<k; col++) {
            idx = row*k + col;
            h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
        }

    }
    for(row=0; row<k; row++) {  //matrix B initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
        }

    }
    for(row=0; row<m; row++) {  //matrix C initialization
        for(col=0; col<n; col++) {
            idx = row*n + col;
            h_C[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
        }

    }

    std::cout << "Test case: m=" << m << ", k=" << k << ", n=" << n << std::endl;

    //DEVICE MEMORY INITIALIZATION
    float *d_A; //matrix A
    float *d_B; //matrix B
    float *d_C; //matrix C

    checkCudaErrors(hipMalloc((void **) &d_A, m*k*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_B, k*n*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_C, m*n*sizeof(float)));

    //copy matrices from the host (CPU) to the device (GPU)
    checkCudaErrors(hipMemcpy(d_A, h_A, m*k*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, k*n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, m*n*sizeof(float), hipMemcpyHostToDevice));

    //CALCULATIONS ON THE CPU - it is useful to check if the calculations on the GPU are correctly made through a comparison of the results.
    float flopCnt = 2.e-6*m*k*n;    //in this case, FLOPS = 2*m*k*n/TIME
    float cpuFlops;
    float gpuFlops;

    //Create the CUDA SDK timer
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    if(exec_cpu[0] == 'y') {
        timer->start();
        cpuMatrixProduct(m, k, n, h_A, h_B, h_C);
        timer->stop();

        cpuFlops = flopCnt / timer->getTime();
        std::cout << "CPU time: " << timer->getTime() << " ms.  GFLOPS: " << cpuFlops << std::endl;
        timer->reset();
    }

    //CALCULATIONS ON THE GPU
    dim3 dimGrid((m-1)/TILE_WIDTH+1, (n-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    timer->start();
    // gpuMatrixProduct<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_C);
    matrixMulti<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, m, k, n);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    gpuFlops = flopCnt / timer->getTime();
    std::cout << "GPU time: " << timer->getTime() << " ms.  GFLOPS: " << gpuFlops << std::endl;

    if(exec_cpu[0] == 'y') {
        //download the resulting matrix d_C from the device and store it in h_C_d.
        checkCudaErrors(hipMemcpy(h_C_d, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost));

        //now let's check if the results are the same.
        float relativeDiff = 0.0f;
        float diff = 0.0f;
        float maxAbs;
        int errCount = 0;

        for(row=0; row<m; row++) {  //comparison between every single entry of h_C with every single entry of h_C_d.
            for(col=0; col<n; col++) {
                idx = row*n + col;

                maxAbs = std::max(std::abs(h_C[idx]), std::abs(h_C_d[idx]));
                if(maxAbs == 0.0)
                    maxAbs = 1.0;
                relativeDiff = std::max(relativeDiff, std:: abs(h_C[idx] - h_C_d[idx])/maxAbs);
                diff = std::max(diff, std::abs(h_C[idx] - h_C_d[idx]));

                if(relativeDiff > 0.001)
                    errCount++;

            }

        }
        //relativeDiff should be as close as possible to unit roundoff.
        //float corresponds to IEEE single precision, so unit roundoff is 1.19e-07.
        std::cout << "Max diff = " << diff << ";    Max relative diff = " << relativeDiff << std::endl;
        std::cout << "Err count = " << errCount << std::endl;
    }
    

    //CLEANING UP
    delete timer;
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_d;

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    return 0;

}
